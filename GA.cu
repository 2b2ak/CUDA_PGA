#include "hip/hip_runtime.h"
#include "Incs.cuh"

using namespace std;

int read_DataSet(string filepath, vector<string> &value)
{
	int LineNumber = 0;
	ifstream file(filepath);
	string str;
	while (getline(file, str))
	{
		if (str.size() > 0)
		{
			value.push_back(str);
		}
		LineNumber++;
	}
	return LineNumber;
}

int main(int argc, char** argv)
{
	hipError_t err = hipSuccess;
	//hipEvent_t Coverage_Start, Coverage_Stop, Reduction_Start, Reduction_Stop, Fitness_Start, Fitness_Stop;

	int Iter, NumofAttr, NumofClass, TestMode;
	int InitRuleRows = 4;

	string DataSetStr, ClassStr;
	
	//float msecTemp = 0.0f;
	int CPU_StartTime, CPU_StopTime;

	float GPUCoverage_msec = 0.0f;
	float GPUReduction_msec = 0.0f;
	float GPUFitness_msec = 0.0f;

	float CPUInitialization_msec = 0.0f;
	float CPUCoverage_msec = 0.0f;
	float CPUAVGCoverage_msec = 0.0f;
	float CPUReduction_msec = 0.0f;
	float CPUFitness_msec = 0.0f;
	float CPUCrossover_msec = 0.0f;
	float CPUMutation_msec = 0.0f;
	float CPUTest_msec = 0.0f;

	/*float CPUCoverage_msec = 0.0f;
	float CPUReduction_msec = 0.0f;
	float CPUFitness_msec = 0.0f;*/

	if (checkCmdLineFlag(argc, (const char **)argv, "Iter"))
	{
		Iter = getCmdLineArgumentInt(argc, (const char **)argv, "Iter");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "Attr"))
	{
		NumofAttr = getCmdLineArgumentInt(argc, (const char **)argv, "Attr");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "Class"))
	{
		NumofClass = getCmdLineArgumentInt(argc, (const char **)argv, "Class");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "RI"))
	{
		if (getCmdLineArgumentInt(argc, (const char **)argv, "RI") >= 4)
		{
			InitRuleRows = getCmdLineArgumentInt(argc, (const char **)argv, "RI");
		}
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "TM"))
	{
		TestMode = getCmdLineArgumentInt(argc, (const char **)argv, "TM");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "TM"))
	{
		TestMode = getCmdLineArgumentInt(argc, (const char **)argv, "TM");
	}
	/*if (checkCmdLineFlag(argc, (const char **)argv, "TRD"))
	{
		DataSetStr = getCmdLineArgumentInt(argc, (const char **)argv, "TRD");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "TRC"))
	{
		ClassStr = getCmdLineArgumentInt(argc, (const char **)argv, "TRC");
	}*/
	DataSetStr = argv[6];
	ClassStr = argv[7];

	float *DataSet;
	int *Class, *d_Class;
	vector<string> DataSet_Row;
	vector<string> Class_Row;
	int NumofRows = read_DataSet(DataSetStr, DataSet_Row);
	if (NumofRows != (read_DataSet(ClassStr, Class_Row)))
	{
		::exit(EXIT_FAILURE);
	}

	int SizeofDataSet = NumofAttr*NumofRows;
	size_t MemSizeDataSet = sizeof(float)*SizeofDataSet;
	size_t ClassMemSize = sizeof(int)*NumofRows;
	hipHostMalloc((void **)&DataSet, MemSizeDataSet);
	hipHostMalloc((void **)&Class, ClassMemSize);
	vector<float> AttrVec;
	vector<int> ClassVec;

	for (int i = 0; i <= NumofRows; i++)
	{
		stringstream StrStm1(DataSet_Row[i]);
		for (float j; StrStm1 >> j;)
		{
			AttrVec.push_back(j);
			if (StrStm1.peek() == ',')
			{
				StrStm1.ignore();
			}
		}
	}

	//printf("%i , %i\n", SizeofDataSet, AttrVec.size());
	for (int i = 0;i < SizeofDataSet;i++)
	{
		DataSet[i] = AttrVec[i];
	}

	if (TestMode == 1)
	{
		::printf("i&a|");
		for (int a = 0;a < NumofAttr;a++)
		{
			::printf("%i|", a);
		}
		::printf("\n");

		for (int i = 0;i < NumofRows;i++)
		{
			::printf(" %i |", i);
			for (int a = 0;a < NumofAttr;a++)
			{
				::printf("%.0f|", DataSet[(i*NumofAttr) + a]);
			}
			::printf("\n");
		}

	}

	for (int i = 0; i <= NumofRows; i++)
	{
		stringstream StrStm2(Class_Row[i]);
		for (int j; StrStm2 >> j;)
		{
			ClassVec.push_back(j);
			if (StrStm2.peek() == ',')
			{
				StrStm2.ignore();
			}
		}
	}

	for (int i = 0;i < NumofRows;i++)
	{
		Class[i] = ClassVec[i];
	}
	hipMalloc((void **)&d_Class, ClassMemSize);

	int *NumClass = new int[NumofClass];

	for (int i = 0;i < NumofClass;i++)
	{
		NumClass[i] = 0;
	}

	NumofEachClass(NumClass, NumofRows, Class);

	for (int i = 0;i < NumofClass;i++)
	{
		::printf("Number of Instances of Class %i = %i\n", i, NumClass[i]);
	}
	::printf("*********************************************\n");

	float *MinValue, *MaxValue;
	int MinMaxSize = NumofAttr;
	size_t MinMax_MemSize = MinMaxSize*sizeof(char);
	hipHostMalloc((void **)&MinValue, MinMax_MemSize);
	hipHostMalloc((void **)&MaxValue, MinMax_MemSize);

	int *Rule_Conditions, *Rule_ActiveAttr, *Coverage;
	float *Rule_LowerBound, *Rule_UpperBound;
	int InitialRuleSize = NumofAttr* InitRuleRows*NumofClass;

	size_t InitialRule_MemSize = InitialRuleSize*sizeof(float);
	size_t InitialRuleCond_MemSize = InitialRuleSize*sizeof(int);
	size_t Coverage_MemSize = InitialRuleSize*NumofRows*sizeof(int);

	hipHostMalloc((void **)&Rule_Conditions, InitialRuleCond_MemSize);
	hipHostMalloc((void **)&Rule_LowerBound, InitialRule_MemSize);
	hipHostMalloc((void **)&Rule_UpperBound, InitialRule_MemSize);
	hipHostMalloc((void **)&Rule_ActiveAttr, InitialRuleCond_MemSize);
	hipHostMalloc((void **)&Coverage, Coverage_MemSize);
	int *CPUCoverage = (int *)malloc(Coverage_MemSize);

	int *d_Rule_Conditions, *d_Rule_ActiveAttr, *d_Coverage;
	float *d_DataSet, *d_Rule_LowerBound, *d_Rule_UpperBound;
	//float *d_MinValue, *d_MaxValue;

	hipMalloc((void **)&d_DataSet, MemSizeDataSet);
	hipMalloc((void **)&d_Rule_Conditions, InitialRuleCond_MemSize);
	hipMalloc((void **)&d_Rule_LowerBound, InitialRule_MemSize);
	hipMalloc((void **)&d_Rule_UpperBound, InitialRule_MemSize);
	hipMalloc((void **)&d_Rule_ActiveAttr, InitialRuleCond_MemSize);
	hipMalloc((void **)&d_Coverage, Coverage_MemSize);

	//Fitness Allocation

	dim3 ThreadsPerBlock(32, 32, 1);
	dim3 BlocksPerGrid(((NumofAttr + 32 - 1) / 32), ((NumofRows + 32 - 1) / 32), (InitRuleRows*NumofClass));

	/*dim3 TPB(NumofAttr, 32);
	dim3 BPG(((NumofAttr + 32 - 1) / 32), ((NumofRows*InitRuleRows*NumofClass + 32 - 1) / 32));*/
	dim3 TPB(32, NumofAttr);
	dim3 BPG(((NumofRows*InitRuleRows*NumofClass + 32 - 1) / 32), ((NumofAttr + 32 - 1) / 32));

	dim3 TPB_Fitness(1, 32, InitRuleRows);
	dim3 BPG_Fitness(1, ((NumofRows + 32 - 1) / 32), NumofClass);

	int *GPU_CoverageResult, *d_GPU_CoverageResult;
	size_t CoverageResult_MemSize = InitRuleRows*NumofClass*NumofRows*sizeof(int);
	hipHostMalloc((void **)&GPU_CoverageResult, CoverageResult_MemSize);
	int *CPU_CoverageMatrix = (int *)malloc(Coverage_MemSize);

	hipMalloc((void **)&d_GPU_CoverageResult, CoverageResult_MemSize);

	int *GPU_CoverageMatrix;
	err = hipHostMalloc((void **)&GPU_CoverageMatrix, Coverage_MemSize);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate vector GPU_CoverageMatrix from device to host (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}

	int *CPU_CoverageResult = (int *)malloc(CoverageResult_MemSize);
	float *GPU_TP, *GPU_FP, *GPU_TN, *GPU_FN;
	float *GPU_Precision, *GPU_TruePositiveRate, *GPU_TrueNegativeRate, *GPU_AccuracyRate, *GPU_Fitness_Value;
	float *d_GPU_TP, *d_GPU_FP, *d_GPU_TN, *d_GPU_FN;
	float *d_GPU_Precision, *d_GPU_TruePositiveRate, *d_GPU_TrueNegativeRate, *d_GPU_AccuracyRate, *d_GPU_Fitness_Value;
	size_t TF_PN_MemSize = InitRuleRows*NumofClass*sizeof(float);
	size_t Fitness_MemSize = InitRuleRows*NumofClass*sizeof(float);
	hipHostMalloc((void **)&GPU_TP, TF_PN_MemSize);
	hipHostMalloc((void **)&GPU_FP, TF_PN_MemSize);
	hipHostMalloc((void **)&GPU_TN, TF_PN_MemSize);
	hipHostMalloc((void **)&GPU_FN, TF_PN_MemSize);
	hipHostMalloc((void **)&GPU_Precision, Fitness_MemSize);
	hipHostMalloc((void **)&GPU_TruePositiveRate, Fitness_MemSize);
	hipHostMalloc((void **)&GPU_TrueNegativeRate, Fitness_MemSize);
	hipHostMalloc((void **)&GPU_AccuracyRate, Fitness_MemSize);
	hipHostMalloc((void **)&GPU_Fitness_Value, Fitness_MemSize);
	float *CPU_TP = (float *)malloc(TF_PN_MemSize);
	float *CPU_FP = (float *)malloc(TF_PN_MemSize);
	float *CPU_TN = (float *)malloc(TF_PN_MemSize);
	float *CPU_FN = (float *)malloc(TF_PN_MemSize);
	float *CPU_Precision = (float *)malloc(Fitness_MemSize);
	float *CPU_TruePositiveRate = (float *)malloc(Fitness_MemSize);
	float *CPU_TrueNegativeRate = (float *)malloc(Fitness_MemSize);
	float *CPU_AccuracyRate = (float *)malloc(Fitness_MemSize);
	float *CPU_Fitness_Value = (float *)malloc(Fitness_MemSize);

	hipMalloc((void **)&d_GPU_TP, TF_PN_MemSize);
	hipMalloc((void **)&d_GPU_FP, TF_PN_MemSize);
	hipMalloc((void **)&d_GPU_TN, TF_PN_MemSize);
	hipMalloc((void **)&d_GPU_FN, TF_PN_MemSize);
	hipMalloc((void **)&d_GPU_Precision, Fitness_MemSize);
	hipMalloc((void **)&d_GPU_TruePositiveRate, Fitness_MemSize);
	hipMalloc((void **)&d_GPU_TrueNegativeRate, Fitness_MemSize);
	hipMalloc((void **)&d_GPU_AccuracyRate, Fitness_MemSize);
	hipMalloc((void **)&d_GPU_Fitness_Value, Fitness_MemSize);

	//Crossover Allocation

	int *SortedFitnessID = (int *)malloc(TF_PN_MemSize);
	float *FitnessSort = (float *)malloc(Fitness_MemSize);

	size_t AvgCoverage_Memsize = InitRuleRows*NumofClass*NumofAttr*sizeof(float);
	float *CPU_AvgCoverage = (float *)malloc(AvgCoverage_Memsize);

	int *Discovered_RC, *Discovered_RA;
	float *Discovered_RL, *Discovered_RU;
	int Discoverded_RI = Iter*NumofAttr*(InitRuleRows / 2)*NumofClass;
	size_t Discoverded_RI_MemSize = Discoverded_RI*sizeof(float);
	size_t Discoverded_RICond_MemSize = Discoverded_RI*sizeof(int);

	hipHostMalloc((void **)&Discovered_RC, Discoverded_RICond_MemSize);
	hipHostMalloc((void **)&Discovered_RL, Discoverded_RI_MemSize);
	hipHostMalloc((void **)&Discovered_RU, Discoverded_RI_MemSize);
	hipHostMalloc((void **)&Discovered_RA, Discoverded_RICond_MemSize);

	bool *MetChromosomes = (bool *)malloc(InitRuleRows*NumofClass*sizeof(bool));

	//Allocations of Testing Phase

	float *LastGenerationError = (float *)malloc(InitRuleRows*NumofClass*sizeof(float));

	//int rnd;
	//for (int i = 0;i < NumofClass;i++)
	//{
	//	for (int j = 0;j < 100;j++)
	//	{
	//		rnd = rand() % NumClass[i];
	//		//printf("iter = %i  |  rnd = %i\n", j, rnd);
	//		RuleInit(DataSet, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, Rule_ActiveAttr, rnd);
	//	}
	//}

	SetMinMax(DataSet, MinValue, MaxValue, NumofAttr, NumofRows);
	for (int i = 0;i < InitialRuleSize*NumofRows;i++)
	{
		Coverage[i] = 8;
		CPUCoverage[i] = 8;
	}
	/*for (int i = 0;i < NumofAttr - 1;i++)
	{
	printf("Min[%i] = %.2f | Max[%i] = %.2f\n", i, MinValue[i], i, MaxValue[i]);
	}*/

	/*dim3 InitThreads(NumofAttr, 3);
	dim3 InitGrid(InitRuleRows, 1);

	RuleInit << <InitGrid, InitThreads >> >(d_Rule_Conditions, d_Rule_LowerBound, d_Rule_UpperBound, d_Rule_ActiveAttr, d_MinValue, d_MaxValue);

	hipMemcpy(Rule_Conditions, d_Rule_Conditions, InitialRule_MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(Rule_LowerBound, d_Rule_LowerBound, InitialRule_MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(Rule_UpperBound, d_Rule_UpperBound, InitialRule_MemSize, hipMemcpyDeviceToHost);
	hipMemcpy(Rule_ActiveAttr, d_Rule_ActiveAttr, InitialRule_MemSize, hipMemcpyDeviceToHost);*/

	/*err = hipEventCreate(&Coverage_Start);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}
	err = hipEventCreate(&Coverage_Stop);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}

	err = hipEventCreate(&Reduction_Start);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}
	err = hipEventCreate(&Reduction_Stop);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}

	err = hipEventCreate(&Fitness_Start);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}
	err = hipEventCreate(&Fitness_Stop);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(err));
		::exit(EXIT_FAILURE);
	}*/

	::printf("Executing Training Phase..\n");
	::printf("Initializing Population..\n");
	CPU_StartTime = clock();

	CPU_InitPopulation_FirstGeneration(Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, MinValue, MaxValue, NumofAttr, InitRuleRows, NumofClass);

	CPU_StopTime = clock();

	::printf("done.\n");

	CPUInitialization_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));
	
	for (int GAIteration = 0;GAIteration < Iter;GAIteration++)
	{
		/*if (GAIteration == 0)
		{
			CPU_InitPopulation_FirstGeneration(Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, MinValue, MaxValue, NumofAttr, InitRuleRows, NumofClass);
		}
		else
		{
			CPU_InitPopulation(Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, MinValue, MaxValue, NumofAttr, InitRuleRows, NumofClass, MetChromosomes);
		}*/

		for (int c = 0;c < NumofClass;c++)
		{
			for (int r = 0;r < InitRuleRows;r++)
			{
				MetChromosomes[(c*InitRuleRows) + r] = false;
			}
		}

		if (TestMode == 2)
		{
			for (int c = 0;c < NumofClass;c++)
			{
				for (int i = 0;i < InitRuleRows;i++)
				{
					for (int j = 0;j < NumofAttr;j++)
					{
						::printf("ActiveAttr[%i] = %i\n", c*InitRuleRows*NumofAttr + (i*NumofAttr) + j, Rule_ActiveAttr[c*InitRuleRows*NumofAttr + (i*NumofAttr) + j]);
						::printf("Condition[%i] = %i\n", c*InitRuleRows*NumofAttr + (i*NumofAttr) + j, Rule_Conditions[c*InitRuleRows*NumofAttr + (i*NumofAttr) + j]);
						::printf("LowerBound[%i] = %.2f\n", c*InitRuleRows*NumofAttr + (i*NumofAttr) + j, Rule_LowerBound[c*InitRuleRows*NumofAttr + (i*NumofAttr) + j]);
						::printf("UpperBound[%i] = %.2f\n", c*InitRuleRows*NumofAttr + (i*NumofAttr) + j, Rule_UpperBound[c*InitRuleRows*NumofAttr + (i*NumofAttr) + j]);
						::printf("*********************************************\n");
					}
				}
			}
		}

		err = hipMemcpy(d_DataSet, DataSet, MemSizeDataSet, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_DataSet (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		err = hipMemcpy(d_Rule_Conditions, Rule_Conditions, InitialRuleCond_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_Rule_Conditions (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		err = hipMemcpy(d_Rule_LowerBound, Rule_LowerBound, InitialRule_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_Rule_LowerBound (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		err = hipMemcpy(d_Rule_UpperBound, Rule_UpperBound, InitialRule_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_Rule_UpperBound (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		err = hipMemcpy(d_Rule_ActiveAttr, Rule_ActiveAttr, InitialRuleCond_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_Rule_ActiveAttr (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		err = hipMemcpy(d_Coverage, Coverage, Coverage_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy device vector d_Coverage (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		/*err = hipEventRecord(Coverage_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();
		
		GPU_CoverageKernel << < BlocksPerGrid, ThreadsPerBlock >> > (d_DataSet, d_Coverage, d_Rule_ActiveAttr, d_Rule_Conditions, d_Rule_LowerBound, d_Rule_UpperBound, NumofRows, NumofAttr);
		err = hipGetLastError();
		hipDeviceSynchronize();
		
		CPU_StopTime = clock();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch GPU_CoverageKernel (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		/*err = hipEventRecord(Coverage_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Coverage_Stop);

		err = hipEventElapsedTime(&msecTemp, Coverage_Start, Coverage_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		GPUCoverage_msec += msecTemp;*/
		GPUCoverage_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		/*err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch GPU_CoverageKernel kernel (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		err = hipMemcpy(Coverage, d_Coverage, Coverage_MemSize, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector Coverage from device to host (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		/*err = hipEventRecord(Coverage_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();

		CPU_CoverageFunction(DataSet, CPUCoverage, Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, NumofAttr, NumofRows, InitRuleRows*NumofClass);

		CPU_StopTime = clock();

		/*err = hipEventRecord(Coverage_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Coverage_Stop);

		err = hipEventElapsedTime(&msecTemp, Coverage_Start, Coverage_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		CPUCoverage_msec += msecTemp;*/
		CPUCoverage_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));


		for (int i = 0;i < InitialRuleSize*NumofRows;i++)
		{
			GPU_CoverageMatrix[i] = Coverage[i];
			CPU_CoverageMatrix[i] = CPUCoverage[i];
		}

		//int cnt = 0;
		//for (int i = 0;i < InitialRuleSize*NumofRows;i++)
		//{
		//	if (Coverage[i] == 1)
		//	{
		//		//printf("Coverage[%i] = %i\n", i, Coverage[i]);
		//		cnt++;
		//	}
		//}
		//printf("# of elements in Coverage = %i\n", InitialRuleSize*NumofRows);
		//printf("# of covered elements = %i\n", cnt);
		//int rndData, rndRule;
		//int sep = 1;
		int cnt = 0;
		if (TestMode == 3)
		{
			//Test Start

			//for (int i = 0;i < 1;i++)
			//{
			//	/*rndData = rand() % NumofRows;
			//	rndRule = rand() % InitRuleRows*NumofClass;*/
			//	rndData = 1;
			//	rndRule = 0;
			//	printf("Selected instance #%i of DataSet and chromosome #%i of intialized rules\n", rndData, rndRule);
			//	for (int j = 0;j < NumofAttr;j++)
			//	{
			//		printf("E = %i |", (rndRule*NumofAttr*NumofRows) + (rndData*NumofAttr) + j);
			//		printf(" D = %.2f |", DataSet[(rndData*NumofAttr) + j]);
			//		printf(" RA = %i |", Rule_ActiveAttr[(rndRule*NumofAttr) + j]);
			//		printf(" RC = %i |", Rule_Conditions[(rndRule*NumofAttr) + j]);
			//		printf(" RL = %.2f |", Rule_LowerBound[(rndRule*NumofAttr) + j]);
			//		printf(" RU = %.2f |", Rule_UpperBound[(rndRule*NumofAttr) + j]);
			//		printf(" C = %i |", CPU_CoverageMatrix[(rndRule*NumofAttr*NumofRows) + (rndData*NumofAttr) + j]);
			//		printf(" C = %i |", GPU_CoverageMatrix[(rndRule*NumofAttr*NumofRows) + (rndData*NumofAttr) + j]);
			//		printf("\n");
			//	}
			//	printf("*********************************************\n");
			//}

			for (int i = 0;i < InitialRuleSize*NumofRows;i++)
			{
				/*if ((sep*NumofAttr) <= i)
				{
					::printf("Passed Row #%i\n", sep);
					sep++;
				}*/
				if (GPU_CoverageMatrix[i] != CPU_CoverageMatrix[i])
				{
					::printf("GPU_CoverageMatrix[%i] != CPU_CoverageMatrix[%i]\n", i, i);
					cnt++;
				}
			}
			::printf("Iter #%i: # of miss-matches in CoverageMatrix = %i\n", GAIteration, cnt);

			//Test Stop
		}

		/*for (int i = 0;i < InitialRuleSize*NumofRows;i++)
		{
			Coverage[i] = CPUCoverage[i];
		}*/
		err = hipMemcpy(d_Coverage, Coverage, Coverage_MemSize, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector Coverage from host to device (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		/*err = hipEventRecord(Reduction_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();

		GPU_CoverageReduction << <BPG, TPB >> >(d_Coverage, NumofAttr, NumofRows, InitRuleRows*NumofClass);
		err = hipGetLastError();
		hipDeviceSynchronize();

		CPU_StopTime = clock();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch GPU_CoverageReduction (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		/*err = hipEventRecord(Reduction_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Reduction_Stop);

		err = hipEventElapsedTime(&msecTemp, Reduction_Start, Reduction_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		GPUReduction_msec += msecTemp;*/
		GPUReduction_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		err = hipMemcpy(Coverage, d_Coverage, Coverage_MemSize, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy vector Coverage from device to host (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		/*err = hipEventRecord(Reduction_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();

		CPU_CoverageReduction(CPUCoverage, NumofAttr, NumofRows, InitRuleRows*NumofClass);

		CPU_StopTime = clock();

		/*err = hipEventRecord(Reduction_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Reduction_Stop);

		err = hipEventElapsedTime(&msecTemp, Reduction_Start, Reduction_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		CPUReduction_msec += msecTemp;*/
		CPUReduction_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		for (int i = 0, j = 0;i < InitialRuleSize*NumofRows;i += NumofAttr, j++)
		{
			GPU_CoverageResult[j] = Coverage[i];
			CPU_CoverageResult[j] = CPUCoverage[i];
		}

		if (TestMode == 4)
		{
			cnt = 0;
			for (int i = 0;i < InitRuleRows*NumofClass*NumofRows;i++)
			{
				if (GPU_CoverageResult[i] != CPU_CoverageResult[i])
				{
					::printf("Passed Row #%i\n", i);
					::printf("GPU_CoverageResult[%i] != CPU_CoverageResult[%i] | %i != %i\n", i, i, GPU_CoverageResult[i], CPU_CoverageResult[i]);
					cnt++;
				}
				/*else
				{
				printf("Passed Row #%i\n", i);
				printf("GPU_CoverageResult[%i] = CPU_CoverageResult[%i] | %i = %i\n", i, i, GPU_CoverageResult[i], CPU_CoverageResult[i]);
				}*/
			}
			::printf("# of miss-matches in CoverageResults = %i\n", cnt);
		}

		for (int i = 0;i < InitRuleRows*NumofClass;i++)
		{
			GPU_TP[i] = 0;
			GPU_FP[i] = 0;
			GPU_TN[i] = 0;
			GPU_FN[i] = 0;
			GPU_Precision[i] = 0;
			GPU_TruePositiveRate[i] = 0;
			GPU_TrueNegativeRate[i] = 0;
			GPU_AccuracyRate[i] = 0;
			GPU_Fitness_Value[i] = 0;

			CPU_TP[i] = 0;
			CPU_FP[i] = 0;
			CPU_TN[i] = 0;
			CPU_FN[i] = 0;
			CPU_Precision[i] = 0;
			CPU_TruePositiveRate[i] = 0;
			CPU_TrueNegativeRate[i] = 0;
			CPU_AccuracyRate[i] = 0;
			CPU_Fitness_Value[i] = 0;
		}

		err = hipMemcpy(d_GPU_CoverageResult, GPU_CoverageResult, CoverageResult_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_Class, Class, ClassMemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_TP, GPU_TP, TF_PN_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_FP, GPU_FP, TF_PN_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_TN, GPU_TN, TF_PN_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_FN, GPU_FN, TF_PN_MemSize, hipMemcpyHostToDevice);

		err = hipMemcpy(d_GPU_Precision, GPU_Precision, Fitness_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_TruePositiveRate, GPU_TruePositiveRate, Fitness_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_TrueNegativeRate, GPU_TrueNegativeRate, Fitness_MemSize, hipMemcpyHostToDevice);
		err = hipMemcpy(d_GPU_Fitness_Value, GPU_Fitness_Value, Fitness_MemSize, hipMemcpyHostToDevice);

		/*err = hipEventRecord(Fitness_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();

		GPU_Fitness << <BPG_Fitness, TPB_Fitness >> >(d_GPU_CoverageResult, d_Class, NumofRows, NumofAttr, NumofClass, InitRuleRows, d_GPU_TP, d_GPU_FP, d_GPU_TN, d_GPU_FN, d_GPU_Precision, d_GPU_TruePositiveRate, d_GPU_TrueNegativeRate, d_GPU_AccuracyRate, d_GPU_Fitness_Value);
		err = hipGetLastError();
		hipDeviceSynchronize();

		CPU_StopTime = clock();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch GPU_Fitness (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}
		/*err = hipEventRecord(Fitness_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Fitness_Stop);

		err = hipEventElapsedTime(&msecTemp, Fitness_Start, Fitness_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		GPUFitness_msec += msecTemp;*/
		GPUFitness_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		err = hipMemcpy(GPU_TP, d_GPU_TP, TF_PN_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_FP, d_GPU_FP, TF_PN_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_TN, d_GPU_TN, TF_PN_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_FN, d_GPU_FN, TF_PN_MemSize, hipMemcpyDeviceToHost);

		err = hipMemcpy(GPU_Precision, d_GPU_Precision, Fitness_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_TruePositiveRate, d_GPU_TruePositiveRate, Fitness_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_TrueNegativeRate, d_GPU_TrueNegativeRate, Fitness_MemSize, hipMemcpyDeviceToHost);
		err = hipMemcpy(GPU_Fitness_Value, d_GPU_Fitness_Value, Fitness_MemSize, hipMemcpyDeviceToHost);

		/*err = hipEventRecord(Fitness_Start, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}*/

		CPU_StartTime = clock();

		CPU_Fitness(CPU_CoverageResult, Class, NumofRows, NumofAttr, NumofClass, InitRuleRows, CPU_TP, CPU_FP, CPU_TN, CPU_FN, CPU_Precision, CPU_TruePositiveRate, CPU_TrueNegativeRate, CPU_AccuracyRate, CPU_Fitness_Value);

		CPU_StopTime = clock();

		/*err = hipEventRecord(Fitness_Stop, NULL);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		err = hipEventSynchronize(Fitness_Stop);

		err = hipEventElapsedTime(&msecTemp, Fitness_Start, Fitness_Stop);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
			::exit(EXIT_FAILURE);
		}

		CPUFitness_msec += msecTemp;*/
 		CPUFitness_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		if (TestMode == 5)
		{
			for (int i = 0;i < InitRuleRows*NumofClass;i++)
			{
				if (GPU_TP[i] != CPU_TP[i])
				{
					::printf("GPU_TP[%i] != CPU_TP[%i] | %.0f != %.0f\n", i, i, GPU_TP[i], CPU_TP[i]);
				}
				else
				{
					::printf("GPU_TP[%i] = CPU_TP[%i] | %.0f = %.0f\n", i, i, GPU_TP[i], CPU_TP[i]);
				}
				if (GPU_FP[i] != CPU_FP[i])
				{
					::printf("GPU_FP[%i] != CPU_FP[%i] | %.0f != %.0f\n", i, i, GPU_FP[i], CPU_FP[i]);
				}
				else
				{
					::printf("GPU_FP[%i] = CPU_FP[%i] | %.0f = %.0f\n", i, i, GPU_FP[i], CPU_FP[i]);
				}
				if (GPU_TN[i] != CPU_TN[i])
				{
					::printf("GPU_TN[%i] != CPU_TN[%i] | %.0f != %.0f\n", i, i, GPU_TN[i], CPU_TN[i]);
				}
				else
				{
					::printf("GPU_TN[%i] = CPU_TN[%i] | %.0f = %.0f\n", i, i, GPU_TN[i], CPU_TN[i]);
				}
				if (GPU_FN[i] != CPU_FN[i])
				{
					::printf("GPU_FN[%i] != CPU_FN[%i] | %.0f != %.0f\n", i, i, GPU_FN[i], CPU_FN[i]);
				}
				else
				{
					::printf("GPU_FN[%i] = CPU_FN[%i] | %.0f = %.0f\n", i, i, GPU_FN[i], CPU_FN[i]);
				}
			}
		}

		if (TestMode == 6)
		{
			for (int i = 0;i < InitRuleRows*NumofClass;i++)
			{
				if (GPU_Precision[i] != CPU_Precision[i])
				{
					::printf("GPU_Precision[%i] != CPU_Precision[%i] | %.2f != %.2f\n", i, i, GPU_Precision[i], CPU_Precision[i]);
				}
				else
				{
					::printf("GPU_Precision[%i] = CPU_Precision[%i] | %.2f = %.2f\n", i, i, GPU_Precision[i], CPU_Precision[i]);
				}
				if (GPU_TruePositiveRate[i] != CPU_TruePositiveRate[i])
				{
					::printf("GPU_TruePositiveRate[%i] != CPU_TruePositiveRate[%i] | %.2f != %.2f\n", i, i, GPU_TruePositiveRate[i], CPU_TruePositiveRate[i]);
				}
				else
				{
					::printf("GPU_TruePositiveRate[%i] = CPU_TruePositiveRate[%i] | %.2f = %.2f\n", i, i, GPU_TruePositiveRate[i], CPU_TruePositiveRate[i]);
				}
				if (GPU_TrueNegativeRate[i] != CPU_TrueNegativeRate[i])
				{
					::printf("GPU_TrueNegativeRate[%i] != CPU_TrueNegativeRate[%i] | %.2f != %.2f\n", i, i, GPU_TrueNegativeRate[i], CPU_TrueNegativeRate[i]);
				}
				else
				{
					::printf("GPU_TrueNegativeRate[%i] = CPU_TrueNegativeRate[%i] | %.2f = %.2f\n", i, i, GPU_TrueNegativeRate[i], CPU_TrueNegativeRate[i]);
				}
				if (GPU_AccuracyRate[i] != CPU_AccuracyRate[i])
				{
					::printf("GPU_AccuracyRate[%i] != CPU_AccuracyRate[%i] | %.2f != %.2f\n", i, i, GPU_AccuracyRate[i], CPU_AccuracyRate[i]);
				}
				else
				{
					::printf("GPU_AccuracyRate[%i] = CPU_AccuracyRate[%i] | %.2f = %.2f\n", i, i, GPU_AccuracyRate[i], CPU_AccuracyRate[i]);
				}
				if (GPU_Fitness_Value[i] != CPU_Fitness_Value[i])
				{
					::printf("GPU_Fitness_Value[%i] != CPU_Fitness_Value[%i] | %.2f != %.2f\n", i, i, GPU_Fitness_Value[i], CPU_Fitness_Value[i]);
				}
				else
				{
					::printf("GPU_Fitness_Value[%i] = CPU_Fitness_Value[%i] | %.2f = %.2f\n", i, i, GPU_Fitness_Value[i], CPU_Fitness_Value[i]);
				}
			}
		}

		for (int i = 0;i < InitRuleRows*NumofClass;i++)
		{
			FitnessSort[i] = CPU_Fitness_Value[i];
		}
		for (int c = 0;c < NumofClass;c++)
		{
			thrust::sort(FitnessSort + (c*InitRuleRows), FitnessSort + ((c + 1)*InitRuleRows));
		}

		CPU_RuleSelection(CPU_Fitness_Value, FitnessSort, SortedFitnessID, NumofClass, InitRuleRows);

		if (TestMode == 7)
		{
			for (int i = 0;i < InitRuleRows*NumofClass;i++)
			{
				::printf("FitnessSort[%i] = %.2f | CPU_Fitness_Value[%i]\n", i, FitnessSort[i], SortedFitnessID[i]);
			}
		}

		for (int i = 0;i < InitRuleRows*NumofClass;i++)
		{
			for (int j = 0;j < NumofAttr;j++)
			{
				CPU_AvgCoverage[(i*NumofAttr) + j] = 0;
			}
		}

		CPU_StartTime = clock();

		CPU_AverageCoverage(CPU_CoverageMatrix, CPU_AvgCoverage, NumClass, NumofClass, InitRuleRows, NumofRows, NumofAttr);

		CPU_StopTime = clock();

		CPUAVGCoverage_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		if (TestMode == 8)
		{
			::printf("Attribute ID =         |");
			for (int i = 0;i < NumofAttr;i++)
			{
				if (i < 10)
				{
					::printf("__%i_|", i);
				}
				else
				{
					::printf("_%i_|", i);
				}
			}
			::printf("\n");
			for (int i = 0;i < InitRuleRows*NumofClass;i++)
			{
				if (i < 10)
				{
					::printf("CPU_AvgCoverage[%i] =   |", i);
				}
				else
				{
					if (i < 100)
					{
						::printf("CPU_AvgCoverage[%i] =  |", i);
					}
				}
				if (i >= 100)
				{
					::printf("CPU_AvgCoverage[%i] = |", i);
				}
				for (int j = 0;j < NumofAttr;j++)
				{
					::printf("%.2f|", CPU_AvgCoverage[(i*NumofAttr) + j]);
				}
				::printf("\n");
			}
		}
		CPU_StartTime = clock();

		CPU_Crossover(CPU_AvgCoverage, SortedFitnessID, MetChromosomes, GAIteration, NumofClass, InitRuleRows, NumofAttr, Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound, Discovered_RA, Discovered_RC, Discovered_RL, Discovered_RU);

		CPU_StopTime = clock();

		CPUCrossover_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

		CPU_StartTime = clock();

		CPU_Mutation(CPU_AvgCoverage, MetChromosomes, MinValue, MaxValue, NumofClass, InitRuleRows, NumofAttr, Rule_ActiveAttr, Rule_Conditions, Rule_LowerBound, Rule_UpperBound);

		CPU_StopTime = clock();

		CPUMutation_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));
	}
	
	::printf("\nTraining is complete.\n");

	DataSetStr = argv[8];
	ClassStr = argv[9];

	/*if (checkCmdLineFlag(argc, (const char **)argv, "TED"))
	{
		DataSetStr = getCmdLineArgumentInt(argc, (const char **)argv, "TED");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "TEC"))
	{
		ClassStr = getCmdLineArgumentInt(argc, (const char **)argv, "TEC");
	}*/

	/*vector<string> TestDataSet_Row;
	vector<string> TestClass_Row;*/
	DataSet_Row.empty();
	DataSet_Row.clear();
	Class_Row.empty();
	Class_Row.clear();
	NumofRows = (read_DataSet(ClassStr, Class_Row));
	//printf("Rows = %i\n", NumofRows);
	if (NumofRows != (read_DataSet(DataSetStr, DataSet_Row)))
	{
		::exit(EXIT_FAILURE);
	}

	float *TestDataSet;
	int *TestClass;
	
	SizeofDataSet = NumofAttr*NumofRows;
	MemSizeDataSet = sizeof(float)*SizeofDataSet;
	ClassMemSize = sizeof(int)*NumofRows;
	hipHostMalloc((void **)&TestDataSet, MemSizeDataSet);
	hipHostMalloc((void **)&TestClass, ClassMemSize);

	/*vector<float> TestAttrVec;
	vector<int> TestClassVec;*/
	AttrVec.empty();
	AttrVec.clear();
	ClassVec.empty();
	ClassVec.clear();
	for (int i = 0; i <= NumofRows; i++)
	{
		stringstream StrStm3(DataSet_Row[i]);
		for (float j; StrStm3 >> j;)
		{
			AttrVec.push_back(j);
			if (StrStm3.peek() == ',')
			{
				StrStm3.ignore();
			}
		}
	}

	//printf("%i , %i\n", SizeofDataSet, AttrVec.size());
	for (int i = 0;i < SizeofDataSet;i++)
	{
		TestDataSet[i] = AttrVec[i];
	}

	for (int i = 0; i <= NumofRows; i++)
	{
		stringstream StrStm4(Class_Row[i]);
		for (int j; StrStm4 >> j;)
		{
			ClassVec.push_back(j);
			if (StrStm4.peek() == ',')
			{
				StrStm4.ignore();
			}
		}
	}

	for (int i = 0;i < NumofRows;i++)
	{
		TestClass[i] = ClassVec[i];
	}

	for (int i = 0;i < NumofClass;i++)
	{
		NumClass[i] = 0;
	}

	NumofEachClass(NumClass, NumofRows, TestClass);

	::printf("\nExecuting Testing Phase..\n");
	if (TestMode == 9)
	{
		::printf("i&a|");
		for (int a = 0;a < NumofAttr;a++)
		{
			::printf("%i|", a);
		}
		::printf("\n");

		for (int i = 0;i < NumofRows;i++)
		{
			::printf(" %i |", i);
			for (int a = 0;a < NumofAttr;a++)
			{
				::printf("%.0f|", TestDataSet[(i*NumofAttr) + a]);
			}
			::printf("\n");
		}

	}
	for (int i = 0;i < NumofClass;i++)
	{
		::printf("Number of Instances of Class %i = %i\n", i, NumClass[i]);
	}
	::printf("*********************************************\n");

	InitRuleRows = InitRuleRows / 2;

	CPU_StartTime = clock();

	TestLastGeneration(LastGenerationError, TestDataSet, TestClass, NumClass, Iter, NumofClass, InitRuleRows, NumofRows, NumofAttr, Discovered_RA, Discovered_RC, Discovered_RL, Discovered_RU);

	CPU_StopTime = clock();

	CPUTest_msec += (CPU_StopTime - CPU_StartTime) / ((double)(CLOCKS_PER_SEC));

	::printf("Chromosome #X of Class #Y        Error\n");
	for (int c = 0;c < NumofClass;c++)
	{
		for (int r = 0;r < InitRuleRows;r++)
		{
			::printf("Chromosome #%i of Class %i        %.2f\n", r, c, LastGenerationError[(c*InitRuleRows) + r]);
		}
	}

	if (TestMode == 10)
	{
		for (int c = 0;c < NumofClass;c++)
		{
			for (int r = 0;r < InitRuleRows;r++)
			{
				::printf("Chromosome #%i of Class #%i\n|", r, c);
				for (int a = 0;a < NumofAttr;a++)
				{
					::printf("|%i: %i_%i_%.2f_%.2f|", a, Discovered_RA[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a], Discovered_RC[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a], Discovered_RL[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a], Discovered_RU[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a]);
					/*::printf("ActiveAttr[%i][%i][%i] = %i\n", c, r, a, Discovered_RA[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a]);
					::printf("Condition[%i][%i][%i] = %i\n", c, r, a, Discovered_RC[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a]);
					::printf("LowerBound[%i][%i][%i] = %.2f\n", c, r, a, Discovered_RL[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a]);
					::printf("UpperBound[%i][%i][%i] = %.2f\n", c, r, a, Discovered_RU[((Iter - 1)*NumofClass*(InitRuleRows / 2)*NumofAttr) + (c*InitRuleRows*NumofAttr) + (r*NumofAttr) + a]);*/
				}
				::printf("|\n*********************************************\n");
			}
		}
	}
	::printf("Analysis of GPU execution time:\n");
	::printf("Execution time of CoverageKernel = %.2f msec\n", (((float)GPUCoverage_msec / Iter) * 1000));
	::printf("Execution time of ReductionKernel = %.2f msec\n", (((float)GPUReduction_msec / Iter) * 1000));
	::printf("Execution time of FitnessKernel = %.2f msec\n", (((float)GPUFitness_msec / Iter) * 1000));
	::printf("*********************************************\n");
	::printf("Analysis of CPU execution time:\n");
	::printf("Execution time of CPU_Initialization = %.2f msec\n", (((float)CPUInitialization_msec / Iter) * 1000));
	::printf("Execution time of CPU_Coverage = %.2f msec\n", (((float)CPUCoverage_msec / Iter) * 1000));
	::printf("Execution time of CPU_Reduction = %.2f msec\n", (((float)CPUReduction_msec / Iter) * 1000));
	::printf("Execution time of CPU_Fitness = %.2f msec\n", (((float)CPUFitness_msec / Iter) * 1000));
	::printf("Execution time of CPU_AVGCoverage = %.2f msec\n", (((float)CPUAVGCoverage_msec / Iter) * 1000));
	::printf("Execution time of CPU_Crossover = %.4f msec\n", (((float)CPUCrossover_msec / Iter) * 1000));
	::printf("Execution time of CPU_Mutation = %.4f msec\n", (((float)CPUMutation_msec / Iter) * 1000));
	::printf("Execution time of CPU_Test = %.2f msec\n", (((float)CPUTest_msec / Iter) * 1000));
	::printf("*********************************************\n");
	::printf("Speedup = (GPU time / CPU time):\n");
	::printf("CoverageKernel Speedup = %.2f\n", ((float)CPUCoverage_msec / GPUCoverage_msec));
	::printf("ReductionKernel Speedup = %.2f\n", ((float)CPUReduction_msec / GPUReduction_msec));
	::printf("FitnessKernel Speedup = %.2f\n", ((float)CPUFitness_msec / GPUFitness_msec));

	//Destroy CUDA events

	/*hipEventDestroy(Coverage_Start);
	hipEventDestroy(Coverage_Stop);
	hipEventDestroy(Reduction_Start);
	hipEventDestroy(Reduction_Stop);
	hipEventDestroy(Fitness_Start);
	hipEventDestroy(Fitness_Stop);*/

	//Clear Device Memory
	
	hipFree(d_DataSet);
	hipFree(d_Coverage);
	hipFree(d_Rule_Conditions);
	hipFree(d_Rule_LowerBound);
	hipFree(d_Rule_UpperBound);
	hipFree(d_Rule_ActiveAttr);
	hipFree(d_GPU_TP);
	hipFree(d_GPU_FP);
	hipFree(d_GPU_TN);
	hipFree(d_GPU_FN);
	hipFree(d_GPU_Precision);
	hipFree(d_GPU_TruePositiveRate);
	hipFree(d_GPU_TrueNegativeRate);
	hipFree(d_GPU_AccuracyRate);
	hipFree(d_GPU_Fitness_Value);
	
	//Clear Host Memory
	hipHostFree(DataSet);
	hipHostFree(Coverage);
	hipHostFree(MinValue);
	hipHostFree(MaxValue);
	hipHostFree(GPU_CoverageMatrix);
	hipHostFree(GPU_CoverageResult);
	hipHostFree(Rule_Conditions);
	hipHostFree(Rule_LowerBound);
	hipHostFree(Rule_UpperBound);
	hipHostFree(Rule_ActiveAttr);
	hipHostFree(GPU_TP);
	hipHostFree(GPU_FP);
	hipHostFree(GPU_TN);
	hipHostFree(GPU_FN);
	hipHostFree(GPU_Precision);
	hipHostFree(GPU_TruePositiveRate);
	hipHostFree(GPU_TrueNegativeRate);
	hipHostFree(GPU_AccuracyRate);
	hipHostFree(GPU_Fitness_Value);
	hipHostFree(TestDataSet);
	hipHostFree(TestClass);

	/*free(CPU_newRA);
	free(CPU_newRC);
	free(CPU_newRL);
	free(CPU_newRU);*/
	::free(CPUCoverage);
	::free(CPU_CoverageMatrix);
	::free(CPU_CoverageResult);
	::free(CPU_TP);
	::free(CPU_FP);
	::free(CPU_TN);
	::free(CPU_FN);
	::free(CPU_Precision);
	::free(CPU_TruePositiveRate);
	::free(CPU_TrueNegativeRate);
	::free(CPU_AccuracyRate);
	::free(CPU_Fitness_Value);
	::free(SortedFitnessID);
	::free(FitnessSort);
	::free(CPU_AvgCoverage);
	::free(MetChromosomes);
	::free(LastGenerationError);
	
	//Exit GA
	::exit(EXIT_SUCCESS);
}

